#include <stdio.h>
#include <hip/hip_runtime.h>


int main(){
    int cudadevice;
    struct hipDeviceProp_t prop;
    hipGetDevice(&cudadevice);
    hipGetDeviceProperties(&prop, cudadevice);
    int mpc = prop.multiProcessorCount;
    int mtpb = prop.maxThreadsPerBlock;
    int shmsize = prop.sharedMemPerBlock;
    printf("Device %d: number of mulitprocessors %d, max number of threads per block %d, shared memory per block %d \n", cudadevice, mpc, mtpb, shmsize);
    return 0;
}